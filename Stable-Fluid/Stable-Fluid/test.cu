#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <time.h>
#include <thread>

__global__ void add(float* x, float* y, float* z, int n)
{
    // 获取全局索引
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // 步长
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
    {
        z[i] = x[i] * y[i];
    }
}

/*
int main(void)
{
    int N = 1e8;
    int nBytes = N * sizeof(float);

    // 申请托管内存
    float* x, * y, * z1, *z2;
    cudaMallocManaged((void**)&x, nBytes);
    cudaMallocManaged((void**)&y, nBytes);
    cudaMallocManaged((void**)&z1, nBytes);

    // 初始化数据
    for (int i = 0; i < N; ++i)
    {
        x[i] = 11110.0;
        y[i] = 22220.0;
    }

    clock_t start1 = clock();
    for (int i = 0; i < N; i++) {
        z2[i] = x[i] * y[i];
    }
    clock_t end1 = clock();
    clock_t start2 = clock();
    // 定义kernel的执行配置
    dim3 blockSize(256);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
    // 执行kernel
    add << < gridSize, blockSize >> > (x, y, z1, N);
    // 同步device 保证结果能正确访问
    cudaDeviceSynchronize();
    // 检查执行结果
    clock_t end2 = clock();
    
    // output
    double t1 = ((double)(end1 - start1)) / CLOCKS_PER_SEC;
    double t2 = ((double)(end2 - start2)) / CLOCKS_PER_SEC;
    std::cout << "time (CPU): " << t1 * 1000 << " ms" << std::endl;
    std::cout << "time (GPU): " << t2 * 1000 << " ms" << std::endl;

    // 释放内存
    cudaFree(x);
    cudaFree(y);
    cudaFree(z1);
    cudaFree(z2);

    return 0;
}
*/